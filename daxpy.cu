
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

//Test


// Device code: Computes Z = aX + Y
__global__
void daxpy(double a, const double* X, const double* Y,
	int arraySize, double* Z)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < arraySize)
		Z[i] = a * X[i] + Y[i];
}

// Host code
void doTheKernelLaunch(double h_a, double* h_X, double* h_Y,
	int arraySize, double* h_Z)
{
	// Now on with the show...
	size_t size = arraySize * sizeof(double);

	// Allocate vectors in device memory
	double* d_X;
	hipMalloc((void**)&d_X, size);
	double* d_Y;
	hipMalloc((void**)&d_Y, size);
	double* d_Z;
	hipMalloc((void**)&d_Z, size);

	// Copy vectors from host memory to device memory
	hipMemcpy(d_X, h_X, size, hipMemcpyHostToDevice);
	hipMemcpy(d_Y, h_Y, size, hipMemcpyHostToDevice);

	// Invoke kernel
	int threadsPerBlock = 256;
	int blocksPerGrid =
		(arraySize + threadsPerBlock - 1) / threadsPerBlock;
	daxpy<<<blocksPerGrid, threadsPerBlock>>>(h_a, d_X, d_Y, arraySize, d_Z);

	// Copy result from device memory to host memory
	// h_Z will contain the result in host memory
	hipMemcpy(h_Z, d_Z, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(d_X);
	hipFree(d_Y);
	hipFree(d_Z);
}

double* do_daxpy(int arraySize)
{
	double a = 2.0;
	double* X = new double[arraySize];
	double* Y = new double[arraySize];
	double* Z = new double[arraySize];
	for (int i=0 ; i<arraySize ; i++)
	{
		X[i] = 1000.0;
		Y[i] =   10.0;
	}
	doTheKernelLaunch(a, X, Y, arraySize, Z);
	for (int i=0 ; i<arraySize ; i++)
		cout << Z[i] << " = " << a << " * " << X[i] << "  +  " << Y[i] << '\n';
	delete [] X;
	delete [] Y;
	return Z;
}

int main()
{
	// report versions
	int driverVersion, runtimeVersion;
	hipError_t dv = hipDriverGetVersion(&driverVersion);
	hipError_t rv = hipRuntimeGetVersion(&runtimeVersion);
	cout << "Driver version: " << driverVersion << "; Runtime version: "
	     << runtimeVersion << "\n\n";

	double* Z = do_daxpy(20);
	// ...
	delete [] Z;
	return 0;
}
